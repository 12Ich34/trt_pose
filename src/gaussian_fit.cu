#include "hip/hip_runtime.h"
#include "gaussian_fit.h"

#include "matrix_multiply.h"
#include "matrix_index.cuh"
#include "matrix_fill.h"
#include "matrix_solve.h"
#include "matrix_copy.h"


// computes residual and jacobian of gaussian fit centered around index
template<typename T>
__global__ void residual_jacobian_d_kernel(
    uint64_t idx, uint8_t N,
    T *cmap_data, matrix_t cmap_mat,
    T *residual_data, matrix_t residual_mat,
    T *jacobian_data, matrix_t jacobian_mat,
    T *param_data, matrix_t param_mat)
{
  int i_offset = threadIdx.x - N / 2;
  int j_offset = threadIdx.y - N / 2;
  
  int i_peak = matrix_unravel_row_r(&cmap_mat, idx);
  int j_peak = matrix_unravel_col_r(&cmap_mat, idx);

  int i = i_peak + i_offset;
  int j = j_peak + j_offset;

  int residual_row = N * threadIdx.x + threadIdx.y;

  // set jacobian and resiudla to 0 if sample is out of bounds
  if (i < 0 || (uint32_t) i >= cmap_mat.rows || j < 0 || (uint32_t) j >= cmap_mat.cols)
  {
    residual_data[matrix_index_c(&residual_mat, residual_row, 0)] = 0;
    jacobian_data[matrix_index_c(&jacobian_mat, residual_row, 0)] = 0;
    jacobian_data[matrix_index_c(&jacobian_mat, residual_row, 1)] = 0;
    jacobian_data[matrix_index_c(&jacobian_mat, residual_row, 2)] = 0;
    jacobian_data[matrix_index_c(&jacobian_mat, residual_row, 3)] = 0;
    return;
  }

  // compute jacobian and jacobian
  T i_diff = i - param_data[0];
  T j_diff = j - param_data[0];
  T i_diff_2 = i_diff * i_diff;
  T j_diff_2 = j_diff * j_diff;
  T exp_val = exp(-(i_diff_2 + j_diff_2) / (2.0 * param_data[3]));
  T ij_coef = -param_data[2] * exp_val / param_data[3];

  residual_data[matrix_index_c(&residual_mat, residual_row, 0)] = cmap_data[matrix_index_r(&cmap_mat, i, j)] - param_data[2] * exp_val;

  jacobian_data[matrix_index_c(&jacobian_mat, residual_row, 0)] = ij_coef * i_diff;
  jacobian_data[matrix_index_c(&jacobian_mat, residual_row, 1)] = ij_coef * j_diff;
  jacobian_data[matrix_index_c(&jacobian_mat, residual_row, 2)] = -exp_val;
  jacobian_data[matrix_index_c(&jacobian_mat, residual_row, 3)] = ij_coef * (i_diff_2 + j_diff_2) / (2.0 * param_data[3]);

}

// computes residual and jacobian of gaussian fit centered around index
// residual mat should be (NxN)x1
// jacobian should be (NxN)x4
// param data should be 4x1
template<typename T>
void residual_jacobian_d(
    uint64_t idx, uint8_t N,
    T *cmap_data, matrix_t *cmap_mat,
    T *residual_data, matrix_t *residual_mat,
    T *jacobian_data, matrix_t *jacobian_mat,
    T *param_data, matrix_t *param_mat, hipStream_t streamId)
{
  static const dim3 blockDim = { N, N }; // 3x3 pixel window used to appx
  residual_jacobian_d_kernel<<<1, blockDim, 0, streamId>>>(idx, N,
      cmap_data, *cmap_mat,
      residual_data, *residual_mat,
      jacobian_data, *jacobian_mat,
      param_data, *param_mat);
}

template __global__ void residual_jacobian_d_kernel(uint64_t, uint8_t, float *, matrix_t, float *, matrix_t, float*, matrix_t, float*, matrix_t);
template void residual_jacobian_d(uint64_t, uint8_t, float *, matrix_t *, float *, matrix_t *, float*, matrix_t *, float*, matrix_t *, hipStream_t);
