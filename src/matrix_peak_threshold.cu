#include "hip/hip_runtime.h"
#include "matrix_index.h"
#include "matrix_peak_threshold.h"

__global__ void matrix_peak_threshold_atomic_d_kernel(matrix_t m, float *data, float threshold, int *count, int *peaks, int max_count)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  // out of bounds
  if (i >= m.rows || j >= m.cols) {
    return; // pixel out of bounds
  }

  int idx = matrix_index_r(&m, i, j);
  float d = data[idx];

  if ((d < threshold) ||
      (i - 1 >= 0 && data[matrix_index_r(&m, i - 1, j)] > d) ||
      (j - 1 >= 0 && data[matrix_index_r(&m, i, j - 1)] > d) ||
      (i + 1 < m.rows && data[matrix_index_r(&m, i + 1, j)] > d) ||
      (j + 1 < m.cols && data[matrix_index_r(&m, i, j + 1)] > d))
  {
    return; // below threshold or higher neighbor
  }

  int m_count = atomicAdd(count, 1);
  if (m_count < max_count) {
    peaks[m_count] = idx;
  }
}

int matrix_peak_threshold_atomic_d(matrix_t *m, float *data, float threshold, int *count, int *peaks, int max_count, hipStream_t streamId)
{
   dim3 blockDim = { 8, 8 };
  dim3 gridDim = { m->rows / 8 + 1, m->cols / 8 + 1 };
  matrix_peak_threshold_atomic_d_kernel<<<gridDim, blockDim, 0, streamId>>>(*m, data, threshold, count, peaks, max_count);
  return 0;
}

template<typename T>
__global__ void matrix_peak_threshold_mask_d_kernel(matrix_t m, T *data, uint8_t *mask, T threshold)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  // out of bounds
  if (i >= m.rows || j >= m.cols) {
    return;
  }

  int idx = matrix_index_r(&m, i, j);
  T d = data[idx];

  if (d < threshold) {
    mask[idx] = 0;
    return;
  }

  // check if neighbor is greater
  if (i - 1 >= 0 && data[matrix_index_r(&m, i - 1, j)] > d) {
    mask[idx] = 0;
    return;
  }

  if (j - 1 >= 0 && data[matrix_index_r(&m, i, j - 1)] > d) {
    mask[idx] = 0;
    return;
  }

  if (i + 1 < m.rows && data[matrix_index_r(&m, i + 1, j)] > d) {
    mask[idx] = 0;
    return;
  }

  if (j + 1 < m.cols && data[matrix_index_r(&m, i, j + 1)] > d) {
    mask[idx] = 0;
    return;
  }

  mask[idx] = 1;
}

template<typename T>
int matrix_peak_threshold_mask_d(matrix_t *m, T *data, uint8_t *mask, T threshold, hipStream_t streamId)
{
  dim3 blockDim = { 8, 8 };
  dim3 gridDim = { m->rows / 8 + 1, m->cols / 8 + 1 };
  matrix_peak_threshold_mask_d_kernel<<<gridDim, blockDim, 0, streamId>>>(*m, data, mask, threshold);
  return 0;
}

// explicit instantiations

template __global__ void matrix_peak_threshold_mask_d_kernel(matrix_t m, float *data, uint8_t *mask, float threshold);
template int matrix_peak_threshold_mask_d(matrix_t *m, float *data, uint8_t *mask, float threshold, hipStream_t streamId);
