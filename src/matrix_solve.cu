#include "matrix_solve.h"

int matrix_solve_c_workspace_size(
    hipsolverHandle_t handle,
    float *a_data, matrix_t *a_mat)
{
  int workspace_size;
  hipsolverDnSpotrf_bufferSize(handle, HIPBLAS_FILL_MODE_UPPER,
      a_mat->rows, a_data, a_mat->rows, &workspace_size); 
  return workspace_size;
};

// must populate b_mat with identity matrix
// matrix must be symmetric positive-definite
int matrix_solve_c(
    hipsolverHandle_t handle,
    float *a_data, matrix_t *a_mat,
    float *b_data, matrix_t *b_mat, 
    float *workspace, int workspace_size)
{
  int *info;
  hipMalloc(&info, sizeof(int));

  // cholesky factorization
  hipsolverDnSpotrf(handle, HIPBLAS_FILL_MODE_UPPER, a_mat->rows, a_data,
    a_mat->rows, workspace, workspace_size, info); 

  // linear solve
  hipsolverDnSpotrs(handle, HIPBLAS_FILL_MODE_UPPER, a_mat->rows, b_mat->cols,
      a_data, a_mat->rows, b_data, b_mat->rows, info);

  hipFree(info);
  return 0;
}
