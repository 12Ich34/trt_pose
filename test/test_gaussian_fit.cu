#include "gtest/gtest.h"

#include "test_utils.h"

#include "hip/hip_runtime.h"

#include "../src/matrix.h"
#include "../src/matrix_fill.h"
#include "../src/matrix_copy.h"
#include "../src/matrix_peak_threshold.h"
#include "../src/gaussian_fit.h"
#include "../src/gauss_newton.h"


TEST(residual_jacobian, Valid)
{
  matrix_t cmap_mat;
  matrix_set_shape(&cmap_mat, 4, 4);

  // create cmap data
  float cmap_data_h[4 * 4] = {
    0, 0.5, 0, 0,
    0.5, 1.0, 0.5, 0,
    0, 0.5, 0, 0,
    0, 0, 0, 0,
  };
  const int max_idx = 5;
  //float cmap_data_h[4 * 4];
  //matrix_copy_h2h_transpose(&cmap_mat, cmap_data_h_transp, cmap_data_h);

  // copy cmap to device
  float *cmap_data_d;
  matrix_malloc_d(&cmap_mat, &cmap_data_d);
  matrix_copy_h2d(&cmap_mat, cmap_data_h, cmap_data_d);

  // create optimization matrices
  const uint8_t N = 3;
  matrix_t jacobian_mat, residual_mat, param_mat;
  matrix_set_shape(&jacobian_mat, N * N, 4);
  matrix_set_shape(&residual_mat, N * N, 1);
  matrix_set_shape(&param_mat, 4, 1);

  float *jacobian_data_h, *residual_data_h, *param_data_h;
  float *jacobian_data_d, *residual_data_d, *param_data_d;
  matrix_malloc_d(&jacobian_mat, &jacobian_data_d);
  matrix_malloc_d(&residual_mat, &residual_data_d);
  matrix_malloc_d(&param_mat, &param_data_d);

  matrix_malloc_h(&jacobian_mat, &jacobian_data_h);
  matrix_malloc_h(&residual_mat, &residual_data_h);
  matrix_malloc_h(&param_mat, &param_data_h);
  
  // initialize parameters
  param_data_h[0] = 1.0;
  param_data_h[1] = 1.0;
  param_data_h[2] = 1.0;
  param_data_h[3] = 1.0;
  matrix_copy_h2d(&param_mat, param_data_h, param_data_d);

  // true jacobian
  float jacobian_data_true_transp_h[] = {
    0.36787944,  0.36787944, -0.36787944, -0.36787944,
    0.60653066, -0.        , -0.60653066, -0.30326533,
    0.36787944, -0.36787944, -0.36787944, -0.36787944,
    -0.        ,  0.60653066, -0.60653066, -0.30326533,
    -0.        , -0.        , -1.        , -0.        ,
    -0.        , -0.60653066, -0.60653066, -0.30326533,
    -0.36787944,  0.36787944, -0.36787944, -0.36787944,
    -0.60653066, -0.        , -0.60653066, -0.30326533,
    -0.36787944, -0.36787944, -0.36787944, -0.36787944
  };
  float jacobian_data_true_h[N * N * 4];
  matrix_copy_h2h_transpose(&jacobian_mat, jacobian_data_true_transp_h, jacobian_data_true_h);

  // true residual
  float residual_data_true_h[] = {
    -0.36787944, -0.10653066, -0.36787944, -0.10653066,  0.        , -0.10653066, -0.36787944, -0.10653066, -0.36787944
  };

  // compute residual jacobian
  residual_jacobian_d(max_idx, N, cmap_data_d, &cmap_mat, residual_data_d, &residual_mat, jacobian_data_d, &jacobian_mat, param_data_d, &param_mat);

  matrix_copy_d2h(&jacobian_mat, jacobian_data_d, jacobian_data_h);
  matrix_copy_d2h(&residual_mat, residual_data_d, residual_data_h);
  matrix_copy_d2h(&param_mat, param_data_d, param_data_h);

  AllFloatEqual(jacobian_data_true_h, jacobian_data_h, matrix_size(&jacobian_mat));
  AllFloatEqual(residual_data_true_h, residual_data_h, matrix_size(&residual_mat));

  hipFree(cmap_data_d);
  hipFree(jacobian_data_d);
  hipFree(residual_data_d);
  hipFree(param_data_d);

  free(jacobian_data_h);
  free(residual_data_h);
  free(param_data_h);
}

TEST(residual_jacobian, ValidNonCentered)
{
  matrix_t cmap_mat;
  matrix_set_shape(&cmap_mat, 4, 4);

  // create cmap data
  float cmap_data_h[4 * 4] = {
    0.0, 0.3, 0.0, 0.0,
    0.3, 0.7, 0.2, 0.0,
    0.0, 0.3, 0.0, 0.0,
    0.0, 0.0, 0.0, 0.0,
  };
  const int max_idx = 5;
  //float cmap_data_h[4 * 4];
  //matrix_copy_h2h_transpose(&cmap_mat, cmap_data_h_transp, cmap_data_h);

  // copy cmap to device
  float *cmap_data_d;
  matrix_malloc_d(&cmap_mat, &cmap_data_d);
  matrix_copy_h2d(&cmap_mat, cmap_data_h, cmap_data_d);

  // create optimization matrices
  const uint8_t N = 3;
  matrix_t jacobian_mat, residual_mat, param_mat;
  matrix_set_shape(&jacobian_mat, N * N, 4);
  matrix_set_shape(&residual_mat, N * N, 1);
  matrix_set_shape(&param_mat, 4, 1);

  float *jacobian_data_h, *residual_data_h, *param_data_h;
  float *jacobian_data_d, *residual_data_d, *param_data_d;
  matrix_malloc_d(&jacobian_mat, &jacobian_data_d);
  matrix_malloc_d(&residual_mat, &residual_data_d);
  matrix_malloc_d(&param_mat, &param_data_d);

  matrix_malloc_h(&jacobian_mat, &jacobian_data_h);
  matrix_malloc_h(&residual_mat, &residual_data_h);
  matrix_malloc_h(&param_mat, &param_data_h);
  
  // initialize parameters
  param_data_h[0] = 1.0;
  param_data_h[1] = 1.0;
  param_data_h[2] = 1.0;
  param_data_h[3] = 1.0;
  matrix_copy_h2d(&param_mat, param_data_h, param_data_d);

  // true jacobian
  float jacobian_data_true_transp_h[] = {
    0.36787944,  0.36787944, -0.36787944, -0.36787944,
    0.60653066, -0.        , -0.60653066, -0.30326533,
    0.36787944, -0.36787944, -0.36787944, -0.36787944,
    -0.        ,  0.60653066, -0.60653066, -0.30326533,
    -0.        , -0.        , -1.        , -0.        ,
    -0.        , -0.60653066, -0.60653066, -0.30326533,
    -0.36787944,  0.36787944, -0.36787944, -0.36787944,
    -0.60653066, -0.        , -0.60653066, -0.30326533,
    -0.36787944, -0.36787944, -0.36787944, -0.36787944
  };
  float jacobian_data_true_h[N * N * 4];
  matrix_copy_h2h_transpose(&jacobian_mat, jacobian_data_true_transp_h, jacobian_data_true_h);

  // true residual
  float residual_data_true_h[] = {
    -0.36787944, -0.30653066, -0.36787944, -0.30653066, -0.3       , -0.40653066, -0.36787944, -0.30653066, -0.36787944
  };

  // compute residual jacobian
  residual_jacobian_d(max_idx, N, cmap_data_d, &cmap_mat, residual_data_d, &residual_mat, jacobian_data_d, &jacobian_mat, param_data_d, &param_mat);

  matrix_copy_d2h(&jacobian_mat, jacobian_data_d, jacobian_data_h);
  matrix_copy_d2h(&residual_mat, residual_data_d, residual_data_h);
  matrix_copy_d2h(&param_mat, param_data_d, param_data_h);

  AllFloatEqual(jacobian_data_true_h, jacobian_data_h, matrix_size(&jacobian_mat));
  AllFloatEqual(residual_data_true_h, residual_data_h, matrix_size(&residual_mat));

  hipFree(cmap_data_d);
  hipFree(jacobian_data_d);
  hipFree(residual_data_d);
  hipFree(param_data_d);

  free(jacobian_data_h);
  free(residual_data_h);
  free(param_data_h);
}

TEST(gauss_newton_step, Valid) {
  matrix_t cmap_mat;
  matrix_set_shape(&cmap_mat, 4, 4);

  // create cmap data
  float cmap_data_h[4 * 4] = {
    0.0, 0.3, 0.0, 0.0,
    0.3, 0.7, 0.2, 0.0,
    0.0, 0.3, 0.0, 0.0,
    0.0, 0.0, 0.0, 0.0,
  };
  const int max_idx = 5;
  //float cmap_data_h[4 * 4];
  //matrix_copy_h2h_transpose(&cmap_mat, cmap_data_h_transp, cmap_data_h);

  // copy cmap to device
  float *cmap_data_d;
  matrix_malloc_d(&cmap_mat, &cmap_data_d);
  matrix_copy_h2d(&cmap_mat, cmap_data_h, cmap_data_d);

  // create optimization matrices
  const uint8_t N = 3;
  matrix_t jacobian_mat, residual_mat, param_mat;
  matrix_set_shape(&jacobian_mat, N * N, 4);
  matrix_set_shape(&residual_mat, N * N, 1);
  matrix_set_shape(&param_mat, 4, 1);

  float *jacobian_data_h, *residual_data_h, *param_data_h;
  float *jacobian_data_d, *residual_data_d, *param_data_d;
  matrix_malloc_d(&jacobian_mat, &jacobian_data_d);
  matrix_malloc_d(&residual_mat, &residual_data_d);
  matrix_malloc_d(&param_mat, &param_data_d);

  matrix_malloc_h(&jacobian_mat, &jacobian_data_h);
  matrix_malloc_h(&residual_mat, &residual_data_h);
  matrix_malloc_h(&param_mat, &param_data_h);
  
  // initialize parameters
  param_data_h[0] = 1.0;
  param_data_h[1] = 1.0;
  param_data_h[2] = 1.0;
  param_data_h[3] = 1.0;
  matrix_copy_h2d(&param_mat, param_data_h, param_data_d);

  residual_jacobian_d(max_idx, N, cmap_data_d, &cmap_mat, residual_data_d, &residual_mat, jacobian_data_d, &jacobian_mat, param_data_d, &param_mat);

  hipblasHandle_t cublasHandle;
  hipsolverHandle_t cusolverHandle;
  hipblasCreate(&cublasHandle);
  hipsolverDnCreate(&cusolverHandle);

  int workspace_size = gauss_newton_step_workspace_size(&jacobian_mat);
  float *workspace;
  hipMalloc(&workspace, workspace_size);

  gauss_newton_step(cublasHandle, cusolverHandle, residual_data_d, &residual_mat, 
      jacobian_data_d, &jacobian_mat, param_data_d, &param_mat, workspace);

  matrix_copy_d2h(&param_mat, param_data_d, param_data_h);

  float param_data_h_true[] = {
    1.0, 0.9525, 0.7372, 0.3315 
  };

  AllNear(param_data_h_true, param_data_h, matrix_size(&param_mat), 0.01f);

  hipFree(cmap_data_d);
  hipFree(jacobian_data_d);
  hipFree(residual_data_d);
  hipFree(param_data_d);
  hipFree(workspace);

  free(jacobian_data_h);
  free(residual_data_h);
  free(param_data_h);

  hipblasDestroy(cublasHandle);
  hipsolverDnDestroy(cusolverHandle);
}

TEST(residual_jacobian, SecondIterValid)
{
  matrix_t cmap_mat;
  matrix_set_shape(&cmap_mat, 4, 4);

  // create cmap data
  float cmap_data_h[4 * 4] = {
    0, 0.3, 0, 0,
    0.3, 0.7, 0.2, 0,
    0, 0.3, 0, 0,
    0, 0, 0, 0,
  };
  const int max_idx = 5;
  //float cmap_data_h[4 * 4];
  //matrix_copy_h2h_transpose(&cmap_mat, cmap_data_h_transp, cmap_data_h);

  // copy cmap to device
  float *cmap_data_d;
  matrix_malloc_d(&cmap_mat, &cmap_data_d);
  matrix_copy_h2d(&cmap_mat, cmap_data_h, cmap_data_d);

  // create optimization matrices
  const uint8_t N = 3;
  matrix_t jacobian_mat, residual_mat, param_mat;
  matrix_set_shape(&jacobian_mat, N * N, 4);
  matrix_set_shape(&residual_mat, N * N, 1);
  matrix_set_shape(&param_mat, 4, 1);

  float *jacobian_data_h, *residual_data_h, *param_data_h;
  float *jacobian_data_d, *residual_data_d, *param_data_d;
  matrix_malloc_d(&jacobian_mat, &jacobian_data_d);
  matrix_malloc_d(&residual_mat, &residual_data_d);
  matrix_malloc_d(&param_mat, &param_data_d);

  matrix_malloc_h(&jacobian_mat, &jacobian_data_h);
  matrix_malloc_h(&residual_mat, &residual_data_h);
  matrix_malloc_h(&param_mat, &param_data_h);
  
  // initialize parameters
  param_data_h[0] = 1.0;
  param_data_h[1] = 0.95250719;
  param_data_h[2] = 0.73715659;
  param_data_h[3] = 0.33148144;
  matrix_copy_h2d(&param_mat, param_data_h, param_data_d);

  // true jacobian
  float jacobian_data_true_transp_h[] = {
    0.12522296,  0.11927577, -0.05630973, -0.36025242,
    0.49039023, -0.02329001, -0.2205166 , -0.74136329,
    0.09402371, -0.09848916, -0.04228018, -0.29743807,
    -0.        ,  0.53905535, -0.25448639, -0.77448393,
    -0.        , -0.10525696, -0.99660353, -0.00754031,
    -0.        , -0.44511228, -0.19108121, -0.70328509,
    -0.12522296,  0.11927577, -0.05630973, -0.36025242,
    -0.49039023, -0.02329001, -0.2205166 , -0.74136329,
    -0.09402371, -0.09848916, -0.04228018, -0.29743807
  };
  float jacobian_data_true_h[N * N * 4];
  matrix_copy_h2h_transpose(&jacobian_mat, jacobian_data_true_transp_h, jacobian_data_true_h);

  // true residual
  float residual_data_true_h[] = {
    -0.04150909,  0.13744474, -0.03116711,  0.11240368, -0.03465286,
            0.05914322, -0.04150909,  0.13744474, -0.03116711
  };

  // compute residual jacobian
  residual_jacobian_d(max_idx, N, cmap_data_d, &cmap_mat, residual_data_d, &residual_mat, jacobian_data_d, &jacobian_mat, param_data_d, &param_mat);

  matrix_copy_d2h(&jacobian_mat, jacobian_data_d, jacobian_data_h);
  matrix_copy_d2h(&residual_mat, residual_data_d, residual_data_h);
  matrix_copy_d2h(&param_mat, param_data_d, param_data_h);

  AllNear(jacobian_data_true_h, jacobian_data_h, matrix_size(&jacobian_mat), 0.001f);
  AllNear(residual_data_true_h, residual_data_h, matrix_size(&residual_mat), 0.001f);

  hipFree(cmap_data_d);
  hipFree(jacobian_data_d);
  hipFree(residual_data_d);
  hipFree(param_data_d);

  free(jacobian_data_h);
  free(residual_data_h);
  free(param_data_h);
}

#ifndef EXCLUDE_MAIN
int main(int argc, char *argv[])
{
  testing::InitGoogleTest(&argc, argv);
  return RUN_ALL_TESTS();
}
#endif
