#include "gtest/gtest.h"

#include "../src/tensor.h"
#include "../src/tensor_matmul.h"

#include "test_utils.h"

TEST(tensor2_matmul, Valid) {
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  tensor2_t aT, bT, cT;
  tensor2_set_sizes(&aT, 2, 2);
  tensor2_set_sizes(&bT, 2, 2);
  tensor2_set_sizes(&cT, 2, 2);

  float aDh[2 * 2] = {
    1, 2, 
    3, 4
  };

  float bDh[2 * 2] = {
    2, 3,
    4, 5
  };

  float cDh[2 * 2];
  float cDh_true[2 * 2] = {
    10, 13,
    22, 29
  };

  float *aD, *bD, *cD;

  hipMalloc(&aD, sizeof(float) * tensor2_get_size(&aT));
  hipMalloc(&bD, sizeof(float) * tensor2_get_size(&bT));
  hipMalloc(&cD, sizeof(float) * tensor2_get_size(&cT));

  hipMemcpy(aD, aDh, sizeof(float) * tensor2_get_size(&aT), hipMemcpyHostToDevice);
  hipMemcpy(bD, bDh, sizeof(float) * tensor2_get_size(&bT), hipMemcpyHostToDevice);

  tensor2_matmul(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
    aD, &aT, bD, &bT, cD, &cT);

  hipMemcpy(cDh, cD, sizeof(float) * tensor2_get_size(&cT), hipMemcpyDeviceToHost);

  AllFloatEqual(cDh, cDh_true, 2 * 2);

  hipblasDestroy(handle);
  hipFree(aD);
  hipFree(bD);
  hipFree(cD);
}

TEST(tensor2_matmul, Valid3x2) {
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  tensor2_t aT, bT, cT;
  tensor2_set_sizes(&aT, 3, 2);
  tensor2_set_sizes(&bT, 2, 3);
  tensor2_set_sizes(&cT, 3, 3);

  float aDh[3 * 2] = {
    1, 2, 
    3, 4,
    5, 6
  };

  float bDh[2 * 3] = {
    2, 3, 4,
    5, 6, 7
  };

  float cDh[3 * 3];
  float cDh_true[3 * 3] = {
    12, 15, 18,
    26, 33, 40,
    40, 51, 62
  };

  float *aD, *bD, *cD;

  hipMalloc(&aD, sizeof(float) * tensor2_get_size(&aT));
  hipMalloc(&bD, sizeof(float) * tensor2_get_size(&bT));
  hipMalloc(&cD, sizeof(float) * tensor2_get_size(&cT));

  hipMemcpy(aD, aDh, sizeof(float) * tensor2_get_size(&aT), hipMemcpyHostToDevice);
  hipMemcpy(bD, bDh, sizeof(float) * tensor2_get_size(&bT), hipMemcpyHostToDevice);

  tensor2_matmul(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
    aD, &aT, bD, &bT, cD, &cT);

  hipMemcpy(cDh, cD, sizeof(float) * tensor2_get_size(&cT), hipMemcpyDeviceToHost);

  AllFloatEqual(cDh, cDh_true, 3 * 3);

  hipblasDestroy(handle);
  hipFree(aD);
  hipFree(bD);
  hipFree(cD);
}

TEST(tensor2_matmul, ValidTranspose3x2) {
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  tensor2_t aT, bT, cT;
  tensor2_set_sizes(&aT, 3, 2);
  tensor2_set_sizes(&bT, 2, 3);
  tensor2_set_sizes(&cT, 3, 3);

  float aDh[3 * 2] = {
    1, 2, 
    3, 4,
    5, 6
  };

  float bDh[2 * 3] = {
    2, 3, 4,
    5, 6, 7
  };

  float cDh[3 * 3];
  float cDh_true[3 * 3] = {
    12, 26, 40,
    15, 33, 51,
    18, 40, 62
  };

  float *aD, *bD, *cD;

  hipMalloc(&aD, sizeof(float) * tensor2_get_size(&aT));
  hipMalloc(&bD, sizeof(float) * tensor2_get_size(&bT));
  hipMalloc(&cD, sizeof(float) * tensor2_get_size(&cT));

  hipMemcpy(aD, aDh, sizeof(float) * tensor2_get_size(&aT), hipMemcpyHostToDevice);
  hipMemcpy(bD, bDh, sizeof(float) * tensor2_get_size(&bT), hipMemcpyHostToDevice);

  tensor2_matmul(handle, HIPBLAS_OP_T, HIPBLAS_OP_T,
    bD, &bT, aD, &aT, cD, &cT);

  hipMemcpy(cDh, cD, sizeof(float) * tensor2_get_size(&cT), hipMemcpyDeviceToHost);

  AllFloatEqual(cDh, cDh_true, 3 * 3);

  hipblasDestroy(handle);
  hipFree(aD);
  hipFree(bD);
  hipFree(cD);
}

TEST(tensor2_matmul, Valid3) {
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  tensor2_t aT, bT, cT;
  tensor2_set_sizes(&aT, 3, 2);
  tensor2_set_sizes(&bT, 2, 3);
  tensor2_set_sizes(&cT, 2, 2);

  float aDh[3 * 2] = {
    1, 2, 
    3, 4,
    5, 6
  };

  float bDh[2 * 3] = {
    2, 3, 4,
    5, 6, 7
  };

  float cDh[2 * 2];
  float cDh_true[2 * 2] = {
    31, 40,
    58, 76
  };

  float *aD, *bD, *cD;

  hipMalloc(&aD, sizeof(float) * tensor2_get_size(&aT));
  hipMalloc(&bD, sizeof(float) * tensor2_get_size(&bT));
  hipMalloc(&cD, sizeof(float) * tensor2_get_size(&cT));

  hipMemcpy(aD, aDh, sizeof(float) * tensor2_get_size(&aT), hipMemcpyHostToDevice);
  hipMemcpy(bD, bDh, sizeof(float) * tensor2_get_size(&bT), hipMemcpyHostToDevice);

  tensor2_matmul(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
    bD, &bT, aD, &aT, cD, &cT);

  hipMemcpy(cDh, cD, sizeof(float) * tensor2_get_size(&cT), hipMemcpyDeviceToHost);

  AllFloatEqual(cDh, cDh_true, 2 * 2);

  hipblasDestroy(handle);
  hipFree(aD);
  hipFree(bD);
  hipFree(cD);
}

int main(int argc, char *argv[])
{
  testing::InitGoogleTest(&argc, argv);
  return RUN_ALL_TESTS();
}
