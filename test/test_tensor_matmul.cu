#include "gtest/gtest.h"

#include "../src/tensor.h"
#include "../src/tensor_matmul.h"

#include "test_utils.h"

TEST(tensor2_matmul, Valid) {
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  tensor2_t aT, bT, cT;
  tensor2_set_sizes(&aT, 2, 2);
  tensor2_set_sizes(&bT, 2, 2);
  tensor2_set_sizes(&cT, 2, 2);

  float aDh[2 * 2] = {
    1, 2, 
    3, 4
  };

  float bDh[2 * 2] = {
    2, 3,
    4, 5
  };

  float cDh[2 * 2];
  float cDh_true[2 * 2] = {
    10, 13,
    22, 29
  };

  float *aD, *bD, *cD;

  hipMalloc(&aD, sizeof(float) * tensor2_get_size(&aT));
  hipMalloc(&bD, sizeof(float) * tensor2_get_size(&bT));
  hipMalloc(&cD, sizeof(float) * tensor2_get_size(&cT));

  hipMemcpy(aD, aDh, sizeof(float) * tensor2_get_size(&aT), hipMemcpyHostToDevice);
  hipMemcpy(bD, bDh, sizeof(float) * tensor2_get_size(&bT), hipMemcpyHostToDevice);

  tensor2_matmul(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
    aD, &aT, bD, &bT, cD, &cT);

  hipMemcpy(cDh, cD, sizeof(float) * tensor2_get_size(&cT), hipMemcpyDeviceToHost);

  ASSERT_EQ(cDh_true[tensor2_index(&cT, 0, 0)], cDh[tensor2_index(&cT, 0, 0)]);
  ASSERT_EQ(cDh_true[tensor2_index(&cT, 0, 1)], cDh[tensor2_index(&cT, 0, 1)]);
  ASSERT_EQ(cDh_true[tensor2_index(&cT, 1, 1)], cDh[tensor2_index(&cT, 1, 1)]);
  ASSERT_EQ(cDh_true[tensor2_index(&cT, 1, 0)], cDh[tensor2_index(&cT, 1, 0)]);

  hipblasDestroy(handle);
}

int main(int argc, char *argv[])
{
  testing::InitGoogleTest(&argc, argv);
  return RUN_ALL_TESTS();
}
